#include <hip/hip_runtime.h>
#include <stdio.h>

#define HANDLE_ERROR(err) {                                                    \
    if ((err) != hipSuccess) {                                                \
        fprintf(stderr, "CUDA error %s:%d: %d\n", __FILE__, __LINE__, hipGetErrorString(err)); \
    }}

#define IX(i, j) ((i) + (N + 2) * (j))
#define SWAP(x0, x)                                                            \
    {                                                                          \
        float *tmp = x0;                                                       \
        x0 = x;                                                                \
        x = tmp;                                                               \
    }

static float *d_u, *d_v, *d_u0, *d_v0, *d_x, *d_x0;

/* will be executed 1D with at least (N+2)^2 threads */
__global__ void cuda_add_source(int N, float *x, const float *s, float dt) {
    int size = (N + 2) * (N + 2);
    int i = threadIdx.x + blockIdx.x * blockDim.x;

    if (i < size) {
        x[i] += dt * s[i];
    }
}

static void call_cuda_add_source(int N, float *x, const float *s, float dt) {
    int size = (N + 2) * (N + 2);

    const dim3 blockSize(512, 1, 1);
    const dim3 gridSize((size + blockSize.x - 1) / blockSize.x, 1, 1);
    cuda_add_source<<<gridSize, blockSize>>>(N, x, s, dt);
}

static void add_source(int N, float *x, float *s, float dt) {
    int size = (N + 2) * (N + 2);
    for (int i = 0; i < size; i++) {
        x[i] += dt * s[i];
    }
}

/* executed 2D with (N+2)x(N+2) */
/* this will be temporary (can just set bounds in other kernels) */
__global__ void cuda_set_bnd(int N, int b, float *x) {
    int tx = threadIdx.x + blockIdx.x * blockDim.x;
    int ty = threadIdx.y + blockIdx.y * blockDim.y;

    if (tx == 0)
        x[IX(tx, ty)] = b == 1 ? -x[IX(1, ty)] : x[IX(1, ty)];
    if (tx == N + 1)
        x[IX(tx, ty)] = b == 1 ? -x[IX(N, ty)] : x[IX(N, ty)];
    if (ty == 0)
        x[IX(tx, ty)] = b == 2 ? -x[IX(tx, 1)] : x[IX(tx, 1)];
    if (ty == N + 1)
        x[IX(tx, ty)] = b == 2 ? -x[IX(tx, N)] : x[IX(tx, N)];

    __syncthreads();

    if (tx == 0 && ty == 0)
        x[IX(tx, ty)] = 0.5f * (x[IX(1, 0)] + x[IX(0, 1)]);
    if (tx == 0 && ty == N + 1)
        x[IX(tx, ty)] = 0.5f * (x[IX(1, N + 1)] + x[IX(0, N)]);
    if (tx == N + 1 && ty == 0)
        x[IX(tx, ty)] = 0.5f * (x[IX(N, 0)] + x[IX(N + 1, 1)]);
    if (tx == N + 1 && ty == N + 1)
        x[IX(tx, ty)] = 0.5f * (x[IX(N, N + 1)] + x[IX(N + 1, N)]);
}

static void set_bnd(int N, int b, float *x) {
    for (int i = 1; i <= N; i++) {
        x[IX(0, i)]     = b == 1 ? -x[IX(1, i)] : x[IX(1, i)];
        x[IX(N + 1, i)] = b == 1 ? -x[IX(N, i)] : x[IX(N, i)];
        x[IX(i, 0)]     = b == 2 ? -x[IX(i, 1)] : x[IX(i, 1)];
        x[IX(i, N + 1)] = b == 2 ? -x[IX(i, N)] : x[IX(i, N)];
    }

    x[IX(0, 0)]         = 0.5f * (x[IX(1, 0)] + x[IX(0, 1)]);
    x[IX(0, N + 1)]     = 0.5f * (x[IX(1, N + 1)] + x[IX(0, N)]);
    x[IX(N + 1, 0)]     = 0.5f * (x[IX(N, 0)] + x[IX(N + 1, 1)]);
    x[IX(N + 1, N + 1)] = 0.5f * (x[IX(N, N + 1)] + x[IX(N + 1, N)]);
}

/* executed 2D with N x N */
__global__ void cuda_lin_solve(int N, int b, float *x, const float *x0, float a, float c) {
    int tx = threadIdx.x + blockIdx.x * blockDim.x;
    int ty = threadIdx.y + blockIdx.y * blockDim.y;

    // account for NxN --> (N+2)x(N+2)
    int i = tx + 1, j = ty + 1;

    if (tx < N && ty < N) {
        x[IX(i, j)] = (x0[IX(i, j)] + a * (x[IX(i - 1, j)] + x[IX(i + 1, j)] + x[IX(i, j - 1)] + x[IX(i, j + 1)])) / c;
    }
}

static void call_cuda_lin_solve(int N, int b, float *x, const float *x0, float a, float c) {
    for (int k = 0; k < 20; k++) {
        cuda_lin_solve<<<dim3((N+16-1)/16, (N+16-1)/16, 1), dim3(16, 16, 1)>>>(N, b, x, x0, a, c);
        cuda_set_bnd<<<dim3((N+2+16-1)/16, (N+2+16-1)/16, 1), dim3(16, 16, 1)>>>(N, b, x);
    }
}

static void lin_solve(int N, int b, float *x, float *x0, float a, float c) {
    for (int k = 0; k < 20; k++) {
        for (int i = 1; i <= N; i++) {
            for (int j = 1; j <= N; j++) {
                x[IX(i, j)] = (x0[IX(i, j)] + a * (x[IX(i - 1, j)] + x[IX(i + 1, j)] + x[IX(i, j - 1)] + x[IX(i, j + 1)])) / c;
            }
        }
        set_bnd(N, b, x);
    }
}

static void call_cuda_diffuse(int N, int b, float *x, float *x0, float diff, float dt) {
    float a = dt * diff * N * N;
    float c = 1 + 4 * a;

    call_cuda_lin_solve(N, b, x, x0, a, c);
}

static void diffuse(int N, int b, float *x, float *x0, float diff, float dt) {
    float a = dt * diff * N * N;
    lin_solve(N, b, x, x0, a, 1 + 4 * a);
}

/* execute 2D with NxN */
__global__ void cuda_advect(int N, int b, float *d, const float *d0, const float *u, const float *v, float dt) {
    int tx = threadIdx.x + blockIdx.x * blockDim.x;
    int ty = threadIdx.y + blockIdx.y * blockDim.y;

    // account for NxN --> (N+2)x(N+2)
    int i = tx + 1, j = ty + 1;

    float dt0 = dt * N;
    if (tx < N && ty < N) {
        float x = i - dt0 * u[IX(i, j)];
        float y = j - dt0 * v[IX(i, j)];
        x = fminf(fmaxf(x, 0.5f), N + 0.5f);
        y = fminf(fmaxf(y, 0.5f), N + 0.5f);
        int i0 = floorf(x);
        int i1 = i0 + 1;
        int j0 = floorf(y);
        int j1 = j0 + 1;

        float s1 = x - i0;
        float s0 = 1 - s1;
        float t1 = y - j0;
        float t0 = 1 - t1;
        d[IX(i, j)] = s0 * (t0 * d0[IX(i0, j0)] + t1 * d0[IX(i0, j1)])
                        + s1 * (t0 * d0[IX(i1, j0)] + t1 * d0[IX(i1, j1)]);
    }
}

static void call_cuda_advect(int N, int b, float *d, const float *d0, const float *u, const float *v, float dt) {
    cuda_advect<<<dim3((N+16-1)/16, (N+16-1)/16, 1), dim3(16, 16, 1)>>>(N, b, d, d0, u, v, dt);
    cuda_set_bnd<<<dim3((N+2+16-1)/16, (N+2+16-1)/16, 1), dim3(16, 16, 1)>>>(N, b, d);
}

static void advect(int N, int b, float *d, float *d0, float *u, float *v, float dt) {
    float dt0 = dt * N;

    for (int i = 1; i <= N; i++) {
        for (int j = 1; j <= N; j++) {
            float x = i - dt0 * u[IX(i, j)];
            float y = j - dt0 * v[IX(i, j)];
            if (x < 0.5f) {
                x = 0.5f;
            }
            if (x > N + 0.5f) {
                x = N + 0.5f;
            }
            int i0 = (int)x;
            int i1 = i0 + 1;
            if (y < 0.5f) {
                y = 0.5f;
            }
            if (y > N + 0.5f) {
                y = N + 0.5f;
            }
            int j0 = (int)y;
            int j1 = j0 + 1;
            float s1 = x - i0;
            float s0 = 1 - s1;
            float t1 = y - j0;
            float t0 = 1 - t1;
            d[IX(i, j)] = s0 * (t0 * d0[IX(i0, j0)] + t1 * d0[IX(i0, j1)])
                          + s1 * (t0 * d0[IX(i1, j0)] + t1 * d0[IX(i1, j1)]);
        }
    }
    set_bnd(N, b, d);
}

__global__ void cuda_project0(int N, float *div, float *p, const float *u, const float *v) {
    int tx = threadIdx.x + blockIdx.x * blockDim.x;
    int ty = threadIdx.y + blockIdx.y * blockDim.y;

    // account for NxN --> (N+2)x(N+2)
    int i = tx + 1, j = ty + 1;

    if (tx < N && ty < N) {
        div[IX(i, j)] = -0.5f * (u[IX(i + 1, j)] - u[IX(i - 1, j)] + v[IX(i, j + 1)] - v[IX(i, j - 1)]) / N;
        p[IX(i, j)] = 0;
    }
}

__global__ void cuda_project1(int N, float *u, float *v, const float *p) {
    int tx = threadIdx.x + blockIdx.x * blockDim.x;
    int ty = threadIdx.y + blockIdx.y * blockDim.y;

    // account for NxN --> (N+2)x(N+2)
    int i = tx + 1, j = ty + 1;

    if (tx < N && ty < N) {
        u[IX(i, j)] -= 0.5f * N * (p[IX(i + 1, j)] - p[IX(i - 1, j)]);
        v[IX(i, j)] -= 0.5f * N * (p[IX(i, j + 1)] - p[IX(i, j - 1)]);
    }
}

static void call_cuda_project(int N, float *u, float *v, float *p, float *div) {
    cuda_project0<<<dim3((N+16-1)/16, (N+16-1)/16, 1), dim3(16, 16, 1)>>>(N, div, p, u, v);

    cuda_set_bnd<<<dim3((N+2+16-1)/16, (N+2+16-1)/16, 1), dim3(16, 16, 1)>>>(N, 0, div);
    cuda_set_bnd<<<dim3((N+2+16-1)/16, (N+2+16-1)/16, 1), dim3(16, 16, 1)>>>(N, 0, p);

    call_cuda_lin_solve(N, 0, p, div, 1, 4);

    cuda_project1<<<dim3((N+16-1)/16, (N+16-1)/16, 1), dim3(16, 16, 1)>>>(N, u, v, p);

    cuda_set_bnd<<<dim3((N+2+16-1)/16, (N+2+16-1)/16, 1), dim3(16, 16, 1)>>>(N, 1, u);
    cuda_set_bnd<<<dim3((N+2+16-1)/16, (N+2+16-1)/16, 1), dim3(16, 16, 1)>>>(N, 2, v);
}

static void project(int N, float *u, float *v, float *p, float *div) {
    for (int i = 1; i <= N; i++) {
        for (int j = 1; j <= N; j++) {
            div[IX(i, j)] = -0.5f * (u[IX(i + 1, j)] - u[IX(i - 1, j)] + v[IX(i, j + 1)] - v[IX(i, j - 1)]) / N;
            p[IX(i, j)] = 0;
        }
    }
    set_bnd(N, 0, div);
    set_bnd(N, 0, p);

    lin_solve(N, 0, p, div, 1, 4);

    for (int i = 1; i <= N; i++) {
        for (int j = 1; j <= N; j++) {
            u[IX(i, j)] -= 0.5f * N * (p[IX(i + 1, j)] - p[IX(i - 1, j)]);
            v[IX(i, j)] -= 0.5f * N * (p[IX(i, j + 1)] - p[IX(i, j - 1)]);
        }
    }
    set_bnd(N, 1, u);
    set_bnd(N, 2, v);
}

void cuda_dens_step(int N, float *x, float *x0, const float *u, const float *v, float diff, float dt) {
    int size = (N + 2) * (N + 2);
    int mem_size = size * sizeof(float);

    HANDLE_ERROR( hipMemcpy(d_x, x, mem_size, hipMemcpyHostToDevice) );
    HANDLE_ERROR( hipMemcpy(d_x0, x0, mem_size, hipMemcpyHostToDevice) );
    HANDLE_ERROR( hipMemcpy(d_u, u, mem_size, hipMemcpyHostToDevice) );
    HANDLE_ERROR( hipMemcpy(d_v, v, mem_size, hipMemcpyHostToDevice) );

    // add_source(N, x, x0, dt);
    call_cuda_add_source(N, d_x, d_x0, dt);

    // SWAP(u0, u);
    SWAP(d_x0, d_x);

    // diffuse(N, 0, x, x0, diff, dt);
    call_cuda_diffuse(N, 0, d_x, d_x0, diff, dt);

    // SWAP(x0, x);
    SWAP(d_x0, d_x);

    // advect(N, 0, x, x0, u, v, dt);
    call_cuda_advect(N, 0, d_x, d_x0, d_u, d_v, dt);

    HANDLE_ERROR( hipMemcpy(x, d_x, mem_size, hipMemcpyDeviceToHost) );
    HANDLE_ERROR( hipMemcpy(x0, d_x0, mem_size, hipMemcpyDeviceToHost) );
}

void cuda_vel_step(int N, float *u, float *v, float *u0, float *v0, float visc, float dt) {
    int size = (N + 2) * (N + 2);
    int mem_size = size * sizeof(float);

    HANDLE_ERROR( hipMemcpy(d_u, u, mem_size, hipMemcpyHostToDevice) );
    HANDLE_ERROR( hipMemcpy(d_v, v, mem_size, hipMemcpyHostToDevice) );
    HANDLE_ERROR( hipMemcpy(d_u0, u0, mem_size, hipMemcpyHostToDevice) );
    HANDLE_ERROR( hipMemcpy(d_v0, v0, mem_size, hipMemcpyHostToDevice) );

    // add_source(N, u, u0, dt);
    // add_source(N, v, v0, dt);
    call_cuda_add_source(N, d_u, d_u0, dt);
    call_cuda_add_source(N, d_v, d_v0, dt);

    // SWAP(u0, u);
    SWAP(d_u0, d_u);

    // diffuse(N, 1, u, u0, visc, dt);
    call_cuda_diffuse(N, 1, d_u, d_u0, visc, dt);

    // SWAP(v0, v);
    SWAP(d_v0, d_v);

    // diffuse(N, 2, v, v0, visc, dt);
    call_cuda_diffuse(N, 2, d_v, d_v0, visc, dt);

    // project(N, u, v, u0, v0);
    call_cuda_project(N, d_u, d_v, d_u0, d_v0);

    // SWAP(u0, u);
    // SWAP(v0, v);
    SWAP(d_u0, d_u);
    SWAP(d_v0, d_v);

    // advect(N, 1, u, u0, u0, v0, dt);
    // advect(N, 2, v, v0, u0, v0, dt);
    call_cuda_advect(N, 1, d_u, d_u0, d_u0, d_v0, dt);
    call_cuda_advect(N, 2, d_v, d_v0, d_u0, d_v0, dt);

    // project(N, u, v, u0, v0);
    call_cuda_project(N, d_u, d_v, d_u0, d_v0);

    HANDLE_ERROR( hipMemcpy(u, d_u, mem_size, hipMemcpyDeviceToHost) );
    HANDLE_ERROR( hipMemcpy(u0, d_u0, mem_size, hipMemcpyDeviceToHost) );
    HANDLE_ERROR( hipMemcpy(v, d_v, mem_size, hipMemcpyDeviceToHost) );
    HANDLE_ERROR( hipMemcpy(v0, d_v0, mem_size, hipMemcpyDeviceToHost) );
}

void cuda_init(int N) {
    int size = (N + 2) * (N + 2) * sizeof(float);

    hipMalloc((void **) &d_u, size);
    hipMalloc((void **) &d_v, size);
    hipMalloc((void **) &d_u0, size);
    hipMalloc((void **) &d_v0, size);
    hipMalloc((void **) &d_x, size);
    hipMalloc((void **) &d_x0, size);

    hipMemset(&d_u, 0, size);
    hipMemset(&d_v, 0, size);
    hipMemset(&d_u0, 0, size);
    hipMemset(&d_v0, 0, size);
    hipMemset(&d_x, 0, size);
    hipMemset(&d_x0, 0, size);
}

void cuda_cleanup() {
    hipFree(d_u);
    hipFree(d_v);
    hipFree(d_u0);
    hipFree(d_v0);
    hipFree(d_x);
    hipFree(d_x0);
}
